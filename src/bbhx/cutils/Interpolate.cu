#include "hip/hip_runtime.h"
#include "global.h"
#include "constants.h"
#include "Interpolate.hh"

#ifdef __HIPCC__
#include "hipsparse.h"
#else
#include "lapacke.h"
#endif

#define NUM_THREADS_INTERPOLATE 256

// See scipy CubicSpline implementation, it matches that
CUDA_CALLABLE_MEMBER
void prep_splines(int i, int length, int interp_i, int ninterps, int num_intermediates, double *b, double *ud, double *diag, double *ld, double *x, double *y, int numBinAll, int param, int nsub, int sub_i)
{
  double dx1, dx2, d, slope1, slope2;
  int ind0x, ind1x, ind2x, ind0y, ind1y, ind2y, ind_out;

  double xval0, xval1, xval2, yval1;

  // get proper frequency array since it is given once for all modes
  int freqArr_i = int(sub_i / num_intermediates);

  // fill values in spline initial computations
  // get indices into the 1D arrays
  // compute necessary quantities
  // fill the diagonals
  if (i == length - 1)
  {

    ind0y = (param * nsub + sub_i) * length + (length - 3);
    ind1y = (param * nsub + sub_i) * length + (length - 2);
    ind2y = (param * nsub + sub_i) * length + (length - 1);

    ind0x = sub_i * length + (length - 3);
    ind1x = sub_i * length + (length - 2);
    ind2x = sub_i * length + (length - 1);

    ind_out = (param * nsub + sub_i) * length + (length - 1);

    xval0 = x[ind0x];
    xval1 = x[ind1x];
    xval2 = x[ind2x];

    dx1 = xval1 - xval0;
    dx2 = xval2 - xval1;
    d = xval2 - xval0;

    yval1 = y[ind1y];

    slope1 = (yval1 - y[ind0y]) / dx1;
    slope2 = (y[ind2y] - yval1) / dx2;

    b[ind_out] = ((dx2 * dx2 * slope1 +
                   (2 * d + dx2) * dx1 * slope2) /
                  d);
    diag[ind_out] = dx1;
    ld[ind_out] = d;
    ud[ind_out] = 0.0;
  }
  else if (i == 0)
  {

    ind0y = (param * nsub + sub_i) * length + 0;
    ind1y = (param * nsub + sub_i) * length + 1;
    ind2y = (param * nsub + sub_i) * length + 2;

    ind0x = sub_i * length + 0;
    ind1x = sub_i * length + 1;
    ind2x = sub_i * length + 2;

    ind_out = (param * nsub + sub_i) * length + 0;

    xval0 = x[ind0x];
    xval1 = x[ind1x];
    xval2 = x[ind2x];

    dx1 = xval1 - xval0;
    dx2 = xval2 - xval1;
    d = xval2 - xval0;

    yval1 = y[ind1y];

    // amp
    slope1 = (yval1 - y[ind0y]) / dx1;
    slope2 = (y[ind2y] - yval1) / dx2;

    b[ind_out] = ((dx1 + 2 * d) * dx2 * slope1 +
                  dx1 * dx1 * slope2) /
                 d;
    ud[ind_out] = d;
    ld[ind_out] = 0.0;
    diag[ind_out] = dx2;
  }
  else
  {

    ind0y = (param * nsub + sub_i) * length + (i - 1);
    ind1y = (param * nsub + sub_i) * length + (i + 0);
    ind2y = (param * nsub + sub_i) * length + (i + 1);

    ind0x = sub_i * length + (i - 1);
    ind1x = sub_i * length + (i + 0);
    ind2x = sub_i * length + (i + 1);

    ind_out = (param * nsub + sub_i) * length + i;

    xval0 = x[ind0x];
    xval1 = x[ind1x];
    xval2 = x[ind2x];

    dx1 = xval1 - xval0;
    dx2 = xval2 - xval1;

    yval1 = y[ind1y];

    // amp
    slope1 = (yval1 - y[ind0y]) / dx1;
    slope2 = (y[ind2y] - yval1) / dx2;

    b[ind_out] = 3.0 * (dx2 * slope1 + dx1 * slope2);
    diag[ind_out] = 2 * (dx1 + dx2);
    ud[ind_out] = dx1;
    ld[ind_out] = dx2;
  }
}

CUDA_KERNEL
void fill_B(double *freqs_arr, double *y_all, double *B, double *upper_diag, double *diag, double *lower_diag,
            int ninterps, int length, int num_intermediates, int numModes, int numBinAll)
{

  int param = 0;
  int nsub = 0;
  int sub_i = 0;
#ifdef __HIPCC__

  int start1 = blockIdx.x;
  int end1 = ninterps;
  int diff1 = gridDim.x;

#else

  int start1 = 0;
  int end1 = ninterps;
  int diff1 = 1;

#endif
  for (int interp_i = start1;
       interp_i < end1; // 2 for re and im
       interp_i += diff1)
  {

#ifdef __HIPCC__

    int start2 = threadIdx.x;
    int end2 = length;
    int diff2 = blockDim.x;

#else

    int start2 = 0;
    int end2 = length;
    int diff2 = 1;

#endif

    param = int((double)interp_i / (numModes * numBinAll));
    nsub = numModes * numBinAll;
    sub_i = interp_i % (numModes * numBinAll);

    for (int i = start2;
         i < end2;
         i += diff2)
    {

      int lead_ind = interp_i * length;
      prep_splines(i, length, interp_i, ninterps, num_intermediates, B, upper_diag, diag, lower_diag, freqs_arr, y_all, numBinAll, param, nsub, sub_i);
    }
  }
}

/*
CuSparse error checking
*/
#define ERR_NE(X, Y)                                                           \
  do                                                                           \
  {                                                                            \
    if ((X) != (Y))                                                            \
    {                                                                          \
      fprintf(stderr, "Error in %s at %s:%d\n", __func__, __FILE__, __LINE__); \
      exit(-1);                                                                \
    }                                                                          \
  } while (0)

#define CUSPARSE_CALL(X) ERR_NE((X), HIPSPARSE_STATUS_SUCCESS)

// See scipy CubicSpline implementation, it matches that
// this is for solving the banded matrix equation
void interpolate_kern(int m, int n, double *a, double *b, double *c, double *d_in)
{
#ifdef __HIPCC__
  size_t bufferSizeInBytes;

  hipsparseHandle_t handle;
  void *pBuffer;

  CUSPARSE_CALL(hipsparseCreate(&handle));
  CUSPARSE_CALL(hipsparseDgtsv2StridedBatch_bufferSizeExt(handle, m, a, b, c, d_in, n, m, &bufferSizeInBytes));
  gpuErrchk(hipMalloc(&pBuffer, bufferSizeInBytes));

  CUSPARSE_CALL(hipsparseDgtsv2StridedBatch(handle,
                                           m,
                                           a, // dl
                                           b, // diag
                                           c, // du
                                           d_in,
                                           n,
                                           m,
                                           pBuffer));

  CUSPARSE_CALL(hipsparseDestroy(handle));
  gpuErrchk(hipFree(pBuffer));

#else

// use lapack on CPU
#ifdef __USE_OMP__
#pragma omp parallel for
#endif
  for (int j = 0;
       j < n;
       j += 1)
  {
    int info = LAPACKE_dgtsv(LAPACK_COL_MAJOR, m, 1, &a[j * m + 1], &b[j * m], &c[j * m], &d_in[j * m], m);
    // if (info != m) printf("lapack info check: %d\n", info);
  }

#endif
}

// See Scipy CubicSpline for more information
CUDA_CALLABLE_MEMBER
void fill_coefficients(int i, int length, int sub_i, int nsub, double *dydx, double dx, double *y, double *coeff1, double *coeff2, double *coeff3, int param)
{
  double slope, t, dydx_i;

  int ind_i = (param * nsub + sub_i) * length + i;
  int ind_ip1 = (param * nsub + sub_i) * length + (i + 1);

  slope = (y[ind_ip1] - y[ind_i]) / dx;

  dydx_i = dydx[ind_i];

  t = (dydx_i + dydx[ind_ip1] - 2 * slope) / dx;

  coeff1[ind_i] = dydx_i;
  coeff2[ind_i] = (slope - dydx_i) / dx - t;
  coeff3[ind_i] = t / dx;
}

CUDA_KERNEL
void set_spline_constants(double *f_arr, double *y, double *c1, double *c2, double *c3, double *B,
                          int ninterps, int length, int num_intermediates, int numBinAll, int numModes)
{

  double df;
#ifdef __HIPCC__
  int start1 = blockIdx.x;
  int end1 = ninterps;
  int diff1 = gridDim.x;
#else

  int start1 = 0;
  int end1 = ninterps;
  int diff1 = 1;

#endif

  for (int interp_i = start1;
       interp_i < end1; // 2 for re and im
       interp_i += diff1)
  {

    int param = (int)(interp_i / (numModes * numBinAll));
    int nsub = numBinAll * numModes;
    int sub_i = interp_i % (numModes * numBinAll);

    // int freqArr_i = sub_i; // int(sub_i / num_intermediates);

#ifdef __HIPCC__
    int start2 = threadIdx.x;
    int end2 = length - 1;
    int diff2 = blockDim.x;
#else

    int start2 = 0;
    int end2 = length - 1;
    int diff2 = 1;

#endif
    for (int i = start2;
         i < end2;
         i += diff2)
    {

      df = f_arr[sub_i * length + (i + 1)] - f_arr[sub_i * length + i];

      int lead_ind = interp_i * length;
      fill_coefficients(i, length, sub_i, nsub, B, df,
                        y,
                        c1,
                        c2,
                        c3, param);
    }
  }
}

void interpolate(double *freqs, double *propArrays,
                 double *B, double *upper_diag, double *diag, double *lower_diag,
                 int length, int numInterpParams, int numModes, int numBinAll)
{

  int num_intermediates = numModes;
  int ninterps = numModes * numInterpParams * numBinAll;

  int nblocks = std::ceil((ninterps + NUM_THREADS_INTERPOLATE - 1) / NUM_THREADS_INTERPOLATE);

  // these are used for both coefficients and diagonals because they are the same size and
  // this reduces the total memory needed
  double *c1 = upper_diag;
  double *c2 = diag;
  double *c3 = lower_diag;

  // process is fill the B matrix which is banded.
  // solve banded matrix equation for spline coefficients
  // Fill the spline coefficients properly

#ifdef __HIPCC__
  fill_B<<<nblocks, NUM_THREADS_INTERPOLATE>>>(freqs, propArrays, B, upper_diag, diag, lower_diag, ninterps, length, num_intermediates, numModes, numBinAll);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());

  interpolate_kern(length, ninterps, lower_diag, diag, upper_diag, B);

  set_spline_constants<<<nblocks, NUM_THREADS_INTERPOLATE>>>(freqs, propArrays, c1, c2, c3, B,
                                                             ninterps, length, num_intermediates, numBinAll, numModes);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());
#else
  fill_B(freqs, propArrays, B, upper_diag, diag, lower_diag, ninterps, length, num_intermediates, numModes, numBinAll);

  interpolate_kern(length, ninterps, lower_diag, diag, upper_diag, B);

  set_spline_constants(freqs, propArrays, c1, c2, c3, B,
                       ninterps, length, num_intermediates, numBinAll, numModes);
#endif
}
