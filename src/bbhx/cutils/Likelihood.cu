#include "hip/hip_runtime.h"
#include "global.h"
#include "constants.h"
#include "Likelihood.hh"

#ifdef __HIPCC__
#include "hip/hip_complex.h"
#include "hipblas.h"
#else
// #include <gsl/gsl_cblas.h>
#endif


#define  NUM_THREADS_LIKE 256

#define  DATA_BLOCK2 512

#ifdef __HIPCC__

#define NUM_THREADS_PREP 256

#ifdef __HIPCC__
CUDA_KERNEL
void new_hdyn_prep(cmplx *A0_out, cmplx *A1_out, cmplx *B0_out, cmplx *B1_out,
    cmplx *h0_arr, cmplx *data, double *psd, double *f_m_arr, double df, double *f_dense, int *data_index_all, int *noise_index_all,
    int *start_inds_all, int *num_points_seg, int length_f_rel, int num_bin, int data_length, int nchannels)
{

    __shared__ cmplx A0[NUM_THREADS_PREP];
    __shared__ cmplx A1[NUM_THREADS_PREP];
    __shared__ cmplx B0[NUM_THREADS_PREP];
    __shared__ cmplx B1[NUM_THREADS_PREP];

    int tid = threadIdx.x;

    int start_ind, num_points, seg_index, data_index, noise_index;
    int data_ind, noise_ind, template_ind, coefficient_index, ind;
    double Sn, f, f_m;
    cmplx d, h0, h0_conj;
    cmplx A0_tmp, A1_tmp, B0_tmp, B1_tmp, A0_flat, B0_flat;
    for (int bin_i = blockIdx.y; bin_i < num_bin; bin_i += gridDim.y)
    {
        data_index = data_index_all[bin_i];
        noise_index = noise_index_all[bin_i];
        for (int chan_i = blockIdx.z; chan_i < nchannels; chan_i += gridDim.z)
        {
            for (int seg_i = blockIdx.x; seg_i < length_f_rel - 1; seg_i += gridDim.x)
            {
                for (int i = threadIdx.x; i < blockDim.x; i += blockDim.x)
                {
                    A0[threadIdx.x] = 0.0;
                    A1[threadIdx.x] = 0.0;
                    B0[threadIdx.x] = 0.0;
                    B1[threadIdx.x] = 0.0;
                }
                __syncthreads();
                
                seg_index = bin_i * length_f_rel + seg_i + 1;
                coefficient_index = (bin_i * nchannels + chan_i) * length_f_rel + seg_i + 1;
                num_points = num_points_seg[seg_index];
                start_ind = start_inds_all[seg_index];
                f_m = f_m_arr[seg_index];

                A0_tmp = 0.0;
                A1_tmp = 0.0;
                B0_tmp = 0.0;
                B1_tmp = 0.0;

                // printf("%d %d %d %d %d\n", bin_i, chan_i, seg_i, length_f_rel, seg_index);
                for (int i = threadIdx.x; i < num_points; i += blockDim.x)
                {
                    ind = i + start_ind;
                    data_ind = (data_index * nchannels + chan_i) * data_length + ind;
                    noise_ind = (noise_index * nchannels + chan_i) * data_length + ind;
                    template_ind = (bin_i * nchannels + chan_i) * data_length + ind;

                    d = data[data_ind];
                    h0 = h0_arr[template_ind];
                    Sn = psd[noise_ind]; // inverse actually
                    f = f_dense[ind];

                    if ((ind > 400990) & (ind < 401000) & (data_index == 10))
                        printf("HUH: %d %d %d %d %e %e %e %e %e\n", ind, data_index, chan_i, nchannels, d.real(), d.imag(), h0.real(), h0.imag(), Sn);

                    h0_conj = gcmplx::conj(h0);

                    A0_flat = 4. * (h0_conj * d) * Sn * df;

                    B0_flat = 4. * (h0_conj * h0) * Sn * df;

                    A0_tmp += A0_flat;
                    A1_tmp += A0_flat * (f - f_m);
                    B0_tmp += B0_flat;
                    B1_tmp += B0_flat * (f - f_m);

                    // printf("check %e %e %e %e\n", A0_tmp.real(), A1_tmp.real(), B0_tmp.real(), B1_tmp.imag());                       
                }

                A0[threadIdx.x] = A0_tmp;
                A1[threadIdx.x] = A1_tmp;
                B0[threadIdx.x] = B0_tmp;
                B1[threadIdx.x] = B1_tmp;

                // if ((bin_i == 0) && (chan_i == 0) && (seg_i == 60)) printf("check %e %e %e %e\n", A0[threadIdx.x].real(), A1[threadIdx.x].real(), B0[threadIdx.x].real(), B1[threadIdx.x].real());

                __syncthreads();
                for (unsigned int s = 1; s < blockDim.x; s *= 2)
                {
                    if (tid % (2 * s) == 0)
                    {
                        A0[tid] += A0[tid + s];
                        A1[tid] += A1[tid + s];
                        B0[tid] += B0[tid + s];
                        B1[tid] += B1[tid + s];
                    }
                    __syncthreads();
                }
                __syncthreads();

                if (threadIdx.x == 0)
                {
                    A0_out[coefficient_index] = A0[0];
                    A1_out[coefficient_index] = A1[0];
                    B0_out[coefficient_index] = B0[0];
                    B1_out[coefficient_index] = B1[0];
                }
                __syncthreads();
            }
        }
    }
}

void new_hdyn_prep_wrap(cmplx *A0_out, cmplx *A1_out, cmplx *B0_out, cmplx *B1_out,
    cmplx *h0_arr, cmplx *data, double *psd, double *f_m_arr, double df, double *f_dense, int *data_index, int *noise_index,
    int *start_inds_all, int *num_points_seg, int length_f_rel, int num_bin, int data_length, int nchannels)
{
    dim3 grid(length_f_rel - 1, num_bin, nchannels);

    new_hdyn_prep<<<grid, NUM_THREADS_PREP>>>(A0_out, A1_out, B0_out, B1_out,
    h0_arr, data, psd, f_m_arr, df, f_dense, data_index, noise_index,
    start_inds_all, num_points_seg, length_f_rel, num_bin, data_length, nchannels);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

}

CUDA_KERNEL
void new_hdyn_like(cmplx* likeOut1, cmplx* likeOut2,
                    cmplx* templateChannels, cmplx* dataConstants,
                    double* dataFreqsIn, int *constants_index,
                    int numBinAll, int length_f_rel, int nChannels, int num_constants)
{
    
    int tid = threadIdx.x;
    __shared__ cmplx like1[NUM_THREADS_PREP];
    __shared__ cmplx like2[NUM_THREADS_PREP];
    cmplx A0, A1, B0, B1, rb, ra;
    cmplx r1, r0, r1Conj, tempLike1, tempLike2;
    double fb, fa, midFreq, mag_r0;
    int const_i;
    for (int bin_i = blockIdx.x; bin_i < numBinAll; bin_i += gridDim.x)
    {
        like1[tid] = 0.0;
        like2[tid] = 0.0;

        tempLike1 = 0.0;
        tempLike2 = 0.0;

        const_i = constants_index[bin_i];

        // sum all channels and i
        for (int chan_i = 0; chan_i < nChannels; chan_i += 1)
        {
            for (int i = threadIdx.x; i < length_f_rel - 1; i += blockDim.x)
            {
                A0 = dataConstants[((0 * num_constants + const_i) * nChannels + chan_i) * (length_f_rel) + i + 1];
                A1 = dataConstants[((1 * num_constants + const_i) * nChannels + chan_i) * (length_f_rel) + i + 1];
                B0 = dataConstants[((2 * num_constants + const_i) * nChannels + chan_i) * (length_f_rel) + i + 1];
                B1 = dataConstants[((3 * num_constants + const_i) * nChannels + chan_i) * (length_f_rel) + i + 1];

                fb = dataFreqsIn[const_i * length_f_rel + i + 1];
                fa = dataFreqsIn[const_i * length_f_rel + i];
                rb = templateChannels[(bin_i * nChannels + chan_i) * length_f_rel + i + 1];
                ra = templateChannels[(bin_i * nChannels + chan_i) * length_f_rel + i];
                // perform the actual computation

                // slope
                r1 = (rb - ra)/(fb - fa);
                midFreq = (fb + fa)/2.0;

                // intercept
                r0 = rb - r1 * (fb - midFreq);

                r1Conj = gcmplx::conj(r1);

                tempLike1 += A0 * gcmplx::conj(r0) + A1 * r1Conj;

                mag_r0 = gcmplx::abs(r0);
                tempLike2 += B0 * (mag_r0 * mag_r0) + 2. * B1 * gcmplx::real(r0 * r1Conj);
                
                // if (bin_i == 0) printf("%d %d %d %e %e %e %e %e %e %e %e\n", bin_i, chan_i, i, tempLike1.real(), tempLike2.real(), r1.real(), r0.real(), fb, fa, A0.real(), B0.real());
                
            }  
        }

        like1[tid] = tempLike1;
        like2[tid] = tempLike2;

        __syncthreads();
        for (unsigned int s = 1; s < blockDim.x; s *= 2)
        {
            if (tid % (2 * s) == 0)
            {
                like1[tid] += like1[tid + s];
                like2[tid] += like2[tid + s];
            }
            __syncthreads();
        }
        __syncthreads();

        if (threadIdx.x == 0)
        {
            likeOut1[bin_i] = like1[0];
            likeOut2[bin_i] = like2[0];
        }
        __syncthreads();
    }
}


void new_hdyn_like_wrap(cmplx* likeOut1, cmplx* likeOut2,
                    cmplx* templateChannels, cmplx* dataConstants,
                    double* dataFreqsIn, int *constants_index,
                    int numBinAll, int length_f_rel, int nChannels, int num_constants)
{
    new_hdyn_like<<<numBinAll, NUM_THREADS_PREP>>>(likeOut1, likeOut2,
                    templateChannels, dataConstants,
                    dataFreqsIn, constants_index,
                    numBinAll, length_f_rel, nChannels, num_constants);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
}

#endif // __HIPCC__
// special way to run this. Need to separate CPU and GPU for this one
CUDA_KERNEL
void hdynLikelihood(cmplx* likeOut1, cmplx* likeOut2,
                    cmplx* templateChannels, cmplx* dataConstants,
                    double* dataFreqsIn,
                    int numBinAll, int data_length, int nChannels)
{

    // shared memory arrays for heterodyning coefficients
    __shared__ cmplx A0temp[DATA_BLOCK2];
    __shared__ cmplx A1temp[DATA_BLOCK2];
    __shared__ cmplx B0temp[DATA_BLOCK2];
    __shared__ cmplx B1temp[DATA_BLOCK2];
    __shared__ double dataFreqs[DATA_BLOCK2];

    // declare variables
    cmplx A0, A1, B0, B1;

    cmplx trans_complex(0.0, 0.0);
    cmplx prev_trans_complex(0.0, 0.0);
    double prevFreq = 0.0;
    double freq = 0.0;

    int currentStart = 0;

    cmplx r0, r1, r1Conj, tempLike1, tempLike2;
    double mag_r0, midFreq;

    int binNum = threadIdx.x + blockDim.x * blockIdx.x;

    tempLike1 = 0.0;
    tempLike2 = 0.0;
    // loop over channels
    for (int channel = 0; channel < nChannels; channel += 1)
    {
        // need to loop through frequencies and store the in shared memory carefully
        prevFreq = 0.0;
        currentStart = 0;
        while (currentStart < data_length)
        {
            __syncthreads();
            for (int jj = threadIdx.x; jj < DATA_BLOCK2; jj += blockDim.x)
            {
                // load in all the information for this group computation
                if ((jj + currentStart) >= data_length) continue;
                A0temp[jj] = dataConstants[(0 * nChannels + channel) * data_length + currentStart + jj];
                A1temp[jj] = dataConstants[(1 * nChannels + channel) * data_length + currentStart + jj];
                B0temp[jj] = dataConstants[(2 * nChannels + channel) * data_length + currentStart + jj];
                B1temp[jj] = dataConstants[(3 * nChannels + channel) * data_length + currentStart + jj];

                dataFreqs[jj] = dataFreqsIn[currentStart + jj];

            }
            __syncthreads();
            if (binNum < numBinAll)
            {
                for (int jj = 0; jj < DATA_BLOCK2; jj += 1)
                {
                    if ((jj + currentStart) >= data_length) continue;
                    freq = dataFreqs[jj];
                    trans_complex = templateChannels[((jj + currentStart) * nChannels + channel) * numBinAll + binNum];

                    // If we are after the first point
                    if ((prevFreq != 0.0) && (jj + currentStart > 0))
                    {
                        A0 = A0temp[jj]; // constants will need to be aligned with 1..n-1 because there are data_length - 1 bins
                        A1 = A1temp[jj];
                        B0 = B0temp[jj];
                        B1 = B1temp[jj];

                        // perform the actual computation

                        // slope
                        r1 = (trans_complex - prev_trans_complex)/(freq - prevFreq);
                        midFreq = (freq + prevFreq)/2.0;

                        // intercept
                        r0 = trans_complex - r1 * (freq - midFreq);

                        r1Conj = gcmplx::conj(r1);

                        tempLike1 += A0 * gcmplx::conj(r0) + A1 * r1Conj;

                        mag_r0 = gcmplx::abs(r0);
                        tempLike2 += B0 * (mag_r0 * mag_r0) + 2. * B1 * gcmplx::real(r0 * r1Conj);
                    }
                    // each step needs info from the last
                    prev_trans_complex = trans_complex;
                    prevFreq = freq;
                }
            }
            currentStart += DATA_BLOCK2;
        }
    }

    // Fill info
    if (binNum < numBinAll)
    {
        likeOut1[binNum] = tempLike1;
        likeOut2[binNum] = tempLike2;
    }
}

#else

// More straighforward on the CPU
void hdynLikelihood(cmplx* likeOut1, cmplx* likeOut2,
                    cmplx* templateChannels, cmplx* dataConstants,
                    double* dataFreqsIn,
                    int numBinAll, int data_length, int nChannels)
{

    #pragma omp parallel for
    for (int binNum = 0; binNum < numBinAll; binNum += 1)
    {
        cmplx A0, A1, B0, B1;

        cmplx trans_complex(0.0, 0.0);
        cmplx prev_trans_complex(0.0, 0.0);
        double prevFreq = 0.0;
        double freq = 0.0;

        cmplx r0, r1, r1Conj, tempLike1, tempLike2;
        double mag_r0, midFreq;

        tempLike1 = 0.0;
        tempLike2 = 0.0;

        for (int channel = 0; channel < nChannels; channel += 1)
        {
            prevFreq = 0.0;
            for (int jj = 0; jj < data_length - 1; jj += 1)
            {
                A0 = dataConstants[(0 * nChannels + channel) * data_length + jj];
                A1 = dataConstants[(1 * nChannels + channel) * data_length + jj];
                B0 = dataConstants[(2 * nChannels + channel) * data_length + jj];
                B1 = dataConstants[(3 * nChannels + channel) * data_length + jj];

                freq = dataFreqsIn[jj];

                trans_complex = templateChannels[((jj) * nChannels + channel) * numBinAll + binNum];

                if ((prevFreq != 0.0) && (jj > 0))
                {
                    r1 = (trans_complex - prev_trans_complex)/(freq - prevFreq);
                    midFreq = (freq + prevFreq)/2.0;

                    r0 = trans_complex - r1 * (freq - midFreq);

                    r1Conj = gcmplx::conj(r1);

                    tempLike1 += A0 * gcmplx::conj(r0) + A1 * r1Conj;

                    mag_r0 = gcmplx::abs(r0);
                    tempLike2 += B0 * (mag_r0 * mag_r0) + 2. * B1 * gcmplx::real(r0 * r1Conj);
                }

                prev_trans_complex = trans_complex;
                prevFreq = freq;
            }
        }
        likeOut1[binNum] = tempLike1;
        likeOut2[binNum] = tempLike2;
    }
}
#endif


void hdyn(cmplx* likeOut1, cmplx* likeOut2,
                    cmplx* templateChannels, cmplx* dataConstants,
                    double* dataFreqs,
                    int numBinAll, int data_length, int nChannels)
{

    int nblocks4 = std::ceil((numBinAll + NUM_THREADS_LIKE -1)/NUM_THREADS_LIKE);
    #ifdef __HIPCC__
    hdynLikelihood <<<nblocks4, NUM_THREADS_LIKE>>> (likeOut1, likeOut2, templateChannels, dataConstants, dataFreqs, numBinAll, data_length, nChannels);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
    #else
    hdynLikelihood(likeOut1, likeOut2, templateChannels, dataConstants, dataFreqs, numBinAll, data_length, nChannels);
    #endif
}

#ifdef __HIPCC__
__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long* address_as_ull =
                              (unsigned long long*)address;
    unsigned long long old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ void atomicAddComplex(cmplx* a, cmplx b){
  //transform the addresses of real and imag. parts to double pointers
  double *x = (double*)a;
  double *y = x+1;
  //use atomicAdd for double variables
  atomicAddDouble(x, b.real());
  atomicAddDouble(y, b.imag());
}
#endif


#define MAX_LENGTH_F_REL 512

// Not used any more but here in case fast computations of all coefficients are needed
// Needs to be checked
CUDA_KERNEL
void prep_hdyn(cmplx* A0_in, cmplx* A1_in, cmplx* B0_in, cmplx* B1_in, cmplx* d_arr, cmplx* h0_arr, double* S_n_arr, double df, int* bins, double* f_dense, double* f_m_arr, int data_length, int nchannels, int length_f_rel)
{

    CUDA_SHARED cmplx A0_temp[MAX_LENGTH_F_REL];
    CUDA_SHARED cmplx A1_temp[MAX_LENGTH_F_REL];
    CUDA_SHARED cmplx B0_temp[MAX_LENGTH_F_REL];
    CUDA_SHARED cmplx B1_temp[MAX_LENGTH_F_REL];

    int start, increment;
    for (int channel = 0; channel < nchannels; channel += 1)
    {
        CUDA_SYNC_THREADS;

        #ifdef __HIPCC__
        start = threadIdx.x;
        increment = blockDim.x;
        #else
        start = 0;
        increment = 1;
        #pragma omp parallel for
        #endif
        for (int i = start; i < length_f_rel - 1; i += increment)
        {
            A0_temp[i + 1] = 0.0;
            A1_temp[i + 1] = 0.0;
            B0_temp[i + 1] = 0.0;
            B1_temp[i + 1] = 0.0;
        }
        CUDA_SYNC_THREADS;

        #ifdef __HIPCC__
        start = threadIdx.x + blockDim.x * blockIdx.x;
        increment = blockDim.x * gridDim.x;
        #else
        start = 0;
        increment = 1;
        #pragma omp parallel for
        #endif
        for (int i = start; i < data_length; i += increment)
        {
            int bin_ind = bins[i];
            cmplx d = d_arr[channel * data_length + i];
            cmplx h0 = h0_arr[channel * data_length + i];
            double S_n = S_n_arr[channel * data_length + i];
            double f = f_dense[i];

            double f_m = f_m_arr[bin_ind];
            cmplx h0_conj = gcmplx::conj(h0);

            cmplx A0_flat = 4. * (h0_conj * d) / S_n * df;
            cmplx A1_flat = A0_flat * (f - f_m);

            cmplx B0_flat = 4. * (h0_conj * h0) / S_n * df;
            cmplx B1_flat = B0_flat * (f - f_m);
            #ifdef __HIPCC__
            atomicAddComplex(&A0_temp[bin_ind + 1], A0_flat);
            atomicAddComplex(&A1_temp[bin_ind + 1], A1_flat);
            atomicAddComplex(&B0_temp[bin_ind + 1], B0_flat);
            atomicAddComplex(&B1_temp[bin_ind + 1], B1_flat);
            #else
            #pragma omp critical
                A0_temp[bin_ind + 1] += A0_flat;
            #pragma omp critical
                A1_temp[bin_ind + 1] += A1_flat;
            #pragma omp critical
                B0_temp[bin_ind + 1] += B0_flat;
            #pragma omp critical
                B1_temp[bin_ind + 1] += B1_flat;
            #endif

        }

        CUDA_SYNC_THREADS;

        #ifdef __HIPCC__
        start = threadIdx.x;
        increment = blockDim.x;
        #else
        start = 0;
        increment = 1;
        #pragma omp parallel for
        #endif
        for (int i = start; i < length_f_rel - 1; i += increment)
        {
            #ifdef __HIPCC__
            atomicAddComplex(&A0_in[channel * length_f_rel + i + 1], A0_temp[i + 1]);
            atomicAddComplex(&A1_in[channel * length_f_rel + i + 1], A1_temp[i + 1]);
            atomicAddComplex(&B0_in[channel * length_f_rel + i + 1], B0_temp[i + 1]);
            atomicAddComplex(&B1_in[channel * length_f_rel + i + 1], B1_temp[i + 1]);
            #else
            #pragma omp critical
                A0_in[channel * length_f_rel + i + 1] += A0_temp[i + 1];
            #pragma omp critical
                A1_in[channel * length_f_rel + i + 1] += A1_temp[i + 1];
            #pragma omp critical
                B0_in[channel * length_f_rel + i + 1] += B0_temp[i + 1];
            #pragma omp critical
                B1_in[channel * length_f_rel + i + 1] += B1_temp[i + 1];
            #endif
        }
        CUDA_SYNC_THREADS;
    }
}

void prep_hdyn_wrap(cmplx* A0_in, cmplx* A1_in, cmplx* B0_in, cmplx* B1_in, cmplx* d_arr, cmplx* h0_arr, double* S_n_arr, double df, int* bins, double* f_dense, double* f_m_arr, int data_length, int nchannels, int length_f_rel)
{
    #ifdef __HIPCC__
    int nblocks = std::ceil((data_length + NUM_THREADS_LIKE -1)/NUM_THREADS_LIKE);
    prep_hdyn<<<nblocks, NUM_THREADS_LIKE>>>(A0_in, A1_in, B0_in, B1_in, d_arr, h0_arr, S_n_arr, df, bins, f_dense, f_m_arr, data_length, nchannels, length_f_rel);

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
    #else
    prep_hdyn(A0_in, A1_in, B0_in, B1_in, d_arr, h0_arr, S_n_arr, df, bins, f_dense, f_m_arr, data_length, nchannels, length_f_rel);
    #endif
}

// add noise weighting efficiently to template
CUDA_KERNEL
void noiseweight_template(cmplx* templateChannels, double* noise_weight_times_df, int ind_start, int length, int data_stream_length, int nChannels)
{
    int start, increment;
    #ifdef __HIPCC__
    start = threadIdx.x + blockDim.x * blockIdx.x;
    increment = gridDim.x * blockDim.x;
    #else
    start = 0;
    increment = 1;
    #pragma omp parallel for
    #endif
    for (int i = start; i < length; i += increment)
    {
        for (int j = 0; j < nChannels; j+= 1)
        {
            templateChannels[j * length + i] = templateChannels[j * length + i] * noise_weight_times_df[j * data_stream_length + ind_start + i];
        }
    }
}

#define NUM_THREADS_LIKE 256

// compute the likelihood directly
// different for CPU and GPU cause of streams
#ifdef __HIPCC__
void direct_like(cmplx* d_h, cmplx* h_h, cmplx* dataChannels, double* noise_weight_times_df, long* templateChannels_ptrs, int* inds_start, int* ind_lengths, int data_stream_length, int numBinAll, int nChannels, int device)
{
    // initialize everything
    hipStream_t streams[numBinAll];
    hipblasHandle_t handle;

    hipDoubleComplex result_d_h[numBinAll];
    hipDoubleComplex result_h_h[numBinAll];

    hipSetDevice(device);
    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
      printf ("CUBLAS initialization failed\n");
      exit(0);
    }

    // omp over streams
    // TODO: can cause errors
    //#pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        // get information for this template
        int length_bin_i = ind_lengths[bin_i];
        int ind_start = inds_start[bin_i];

        cmplx* templateChannels = (cmplx*) templateChannels_ptrs[bin_i];

        int nblocks = std::ceil((length_bin_i + NUM_THREADS_LIKE -1)/NUM_THREADS_LIKE);
        hipStreamCreate(&streams[bin_i]);

        noiseweight_template
        <<<nblocks, NUM_THREADS_LIKE, 0, streams[bin_i]>>>(templateChannels, noise_weight_times_df, ind_start, length_bin_i, data_stream_length, nChannels);
        hipStreamSynchronize(streams[bin_i]);

        for (int j = 0; j < nChannels; j += 1)
        {
            // setup cublas stream and run compuation in the desired frequency bounds
            double temp_real = 0.0;
            double temp_imag = 0.0;

            // d_h computation
            hipblasSetStream(handle, streams[bin_i]);
            stat = hipblasZdotc(handle, length_bin_i,
                              (hipDoubleComplex*)&dataChannels[j * data_stream_length + ind_start], 1,
                              (hipDoubleComplex*)&templateChannels[j * length_bin_i], 1,
                              &result_d_h[bin_i]);
            hipStreamSynchronize(streams[bin_i]);
            if (stat != HIPBLAS_STATUS_SUCCESS)
            {
                exit(0);
            }

            temp_real = hipCreal(result_d_h[bin_i]);
            temp_imag = hipCimag(result_d_h[bin_i]);
            cmplx temp_d_h(temp_real, temp_imag);
            d_h[bin_i] += 4.0 * temp_d_h;

            // h_h computation
            hipblasSetStream(handle, streams[bin_i]);
            stat = hipblasZdotc(handle, length_bin_i,
                              (hipDoubleComplex*)&templateChannels[j * length_bin_i], 1,
                              (hipDoubleComplex*)&templateChannels[j * length_bin_i], 1,
                              &result_h_h[bin_i]);
            hipStreamSynchronize(streams[bin_i]);
            if (stat != HIPBLAS_STATUS_SUCCESS)
            {
                exit(0);
            }

            temp_real = hipCreal(result_h_h[bin_i]);
            temp_imag = hipCimag(result_h_h[bin_i]);
            cmplx temp_h_h(temp_real, temp_imag);
            h_h[bin_i] += 4.0 * temp_h_h;

        }
    }

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    //#pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        //destroy the streams
        hipStreamDestroy(streams[bin_i]);
    }
    hipblasDestroy(handle);

}

#else

// matmul sub for cblas for backward compatibility
// FORTRAN STYLE COLUMN MAJOR (UGH)
template <typename T>
CUDA_CALLABLE_MEMBER void our_cblas_zdotc(
    int m,
    T *a,
    T *b,
    T *c)
{
    cmplx _out = 0.0;
    for (int i = 0; i < m; i++)
    {
       _out += gcmplx::conj(a[i]) * b[i];
    }
    *c = _out;
}

void direct_like(cmplx* d_h, cmplx* h_h, cmplx* dataChannels, double* noise_weight_times_df, long* templateChannels_ptrs, int* inds_start, int* ind_lengths, int data_stream_length, int numBinAll, int nChannels, int device)
{

    cmplx result_d_h[numBinAll];
    cmplx result_h_h[numBinAll];

    #pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        int length_bin_i = ind_lengths[bin_i];
        int ind_start = inds_start[bin_i];

        cmplx* templateChannels = (cmplx*) templateChannels_ptrs[bin_i];

        noiseweight_template
        (templateChannels, noise_weight_times_df, ind_start, length_bin_i, data_stream_length, nChannels);

        for (int j = 0; j < nChannels; j += 1)
        {

            our_cblas_zdotc(length_bin_i,
                              &dataChannels[j * data_stream_length + ind_start],
                              &templateChannels[j * length_bin_i],
                              &result_d_h[bin_i]);

            d_h[bin_i] += 4.0 * result_d_h[bin_i];

            our_cblas_zdotc(length_bin_i,
                              &templateChannels[j * length_bin_i],
                              &templateChannels[j * length_bin_i],
                              &result_h_h[bin_i]);

            h_h[bin_i] += 4.0 * result_h_h[bin_i];
            //printf("%e %e\n", hipCreal(result_h_h[bin_i]), hipCreal(result_d_h[bin_i]));

        }
    }
}
#endif
