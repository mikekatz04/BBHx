#include "hip/hip_runtime.h"
#include "stdio.h"

#include "global.h"
#include "constants.h"
#include "Response.hh"
#include "Detector.hpp"
#include <iostream>

#define NUM_THREADS_RESPONSE 64

CUDA_CALLABLE_MEMBER
double d_dot_product_1d(double *arr1, double *arr2)
{
    double out = 0.0;
    for (int i = 0; i < 3; i++)
    {
        out += arr1[i] * arr2[i];
    }
    return out;
}

CUDA_CALLABLE_MEMBER
cmplx d_vec_H_vec_product(double *arr1, cmplx *H, double *arr2)
{

    cmplx I(0.0, 1.0);
    cmplx out(0.0, 0.0);
    cmplx trans(0.0, 0.0);
    for (int i = 0; i < 3; i++)
    {
        trans = cmplx(0.0, 0.0);
        for (int j = 0; j < 3; j++)
        {
            trans += (H[i * 3 + j] * arr2[j]);
        }
        out += arr1[i] * trans;
    }
    return out;
}

CUDA_CALLABLE_MEMBER
double d_sinc(double x)
{
    if (x == 0.0)
        return 1.0;
    else
        return sin(x) / x;
}

/* # Single-link response
# 'full' does include the orbital-delay term, 'constellation' does not
 */
CUDA_CALLABLE_MEMBER
d_Gslr_holder d_EvaluateGslr(double t, double f, cmplx *H, double *k, int response, double *p0, Orbits *orbits)
{
    // response == 1 is full, response anything else is constellation
    // Trajectories, p0 used only for the full response
    cmplx I(0.0, 1.0);
    cmplx m_I(0.0, -1.0);
    double alpha = Omega0 * t;
    double c = cos(alpha);
    double s = sin(alpha);
    double a = AU_SI;
    double e = eorbit;

#ifdef __HIPCC__
    CUDA_SHARED double p1L_all[NUM_THREADS_RESPONSE * 3];
    double *p1L = &p1L_all[threadIdx.x * 3];
    CUDA_SHARED double p2L_all[NUM_THREADS_RESPONSE * 3];
    double *p2L = &p2L_all[threadIdx.x * 3];
    CUDA_SHARED double p3L_all[NUM_THREADS_RESPONSE * 3];
    double *p3L = &p3L_all[threadIdx.x * 3];
    CUDA_SHARED double n_all[NUM_THREADS_RESPONSE * 3];
    double *n = &n_all[threadIdx.x * 3];
#else
    double p1L_all[3];
    double *p1L = &p1L_all[0];
    double p2L_all[3];
    double *p2L = &p2L_all[0];
    double p3L_all[3];
    double *p3L = &p3L_all[0];
    double n_all[3];
    double *n = &n_all[0];

#endif
    Vec tmp = orbits->get_pos(t, 1);
    p1L[0] = tmp.x;
    p1L[1] = tmp.y;
    p1L[2] = tmp.z;

    tmp = orbits->get_pos(t, 2);
    p2L[0] = tmp.x;
    p2L[1] = tmp.y;
    p2L[2] = tmp.z;

    tmp = orbits->get_pos(t, 3);
    p3L[0] = tmp.x;
    p3L[1] = tmp.y;
    p3L[2] = tmp.z;

    // n1
    // TODO: need to check this
    tmp = orbits->get_normal_unit_vec(t, 12);
    n[0] = tmp.x;
    n[1] = tmp.y;
    n[2] = tmp.z;

    double kn1 = d_dot_product_1d(k, n);
    cmplx n1Hn1 = d_vec_H_vec_product(n, H, n); // np.dot(n1, np.dot(H, n1))

    // n2
    tmp = orbits->get_normal_unit_vec(t, 23);
    n[0] = tmp.x;
    n[1] = tmp.y;
    n[2] = tmp.z;

    // for (int i = 0; i < 3; i++)
    //     n[i] = n[i] * 1. / 4.;

    double kn2 = d_dot_product_1d(k, n);
    cmplx n2Hn2 = d_vec_H_vec_product(n, H, n); // np.dot(n1, np.dot(H, n1))

    // n3
    tmp = orbits->get_normal_unit_vec(t, 31);
    n[0] = tmp.x;
    n[1] = tmp.y;
    n[2] = tmp.z;

    // for (int i = 0; i < 3; i++)
    //     n[i] = n[i] * 1. / 4.;

    double kn3 = d_dot_product_1d(k, n);
    cmplx n3Hn3 = d_vec_H_vec_product(n, H, n); // np.dot(n1, np.dot(H, n1))

    // # Compute intermediate scalar products
    // t scalar case

    double temp1 = p1L[0] + p2L[0];
    double temp2 = p1L[1] + p2L[1];
    double temp3 = p1L[2] + p2L[2];
    double temp4 = p2L[0] + p3L[0];
    double temp5 = p2L[1] + p3L[1];
    double temp6 = p2L[2] + p3L[2];
    double temp7 = p3L[0] + p1L[0];
    double temp8 = p3L[1] + p1L[1];
    double temp9 = p3L[2] + p1L[2];

    p1L[0] = temp1;
    p1L[1] = temp2;
    p1L[2] = temp3; // now p1L_plus_p2L -> p1L
    p2L[0] = temp4;
    p2L[1] = temp5;
    p2L[2] = temp6; // now p2L_plus_p3L -> p2L
    p3L[0] = temp7;
    p3L[1] = temp8;
    p3L[2] = temp9; // now p3L_plus_p1L -> p3L

    double kp1Lp2L = d_dot_product_1d(k, p1L);
    double kp2Lp3L = d_dot_product_1d(k, p2L);
    double kp3Lp1L = d_dot_product_1d(k, p3L);
    double kp0 = d_dot_product_1d(k, p0);

    // # Prefactors - projections are either scalars or vectors
    cmplx factorcexp0;
    if (response == 1)
        factorcexp0 = gcmplx::exp(I * 2. * PI * f / C_SI * kp0); // I*2.*PI*f/C_SI * kp0
    else
        factorcexp0 = cmplx(1.0, 0.0);
    double prefactor = PI * f * orbits->armlength / C_SI;

    cmplx factorcexp12 = gcmplx::exp(I * prefactor * (1. + kp1Lp2L / orbits->armlength)); // prefactor * (1.+kp1Lp2L/orbits->armlength)
    cmplx factorcexp23 = gcmplx::exp(I * prefactor * (1. + kp2Lp3L / orbits->armlength)); // prefactor * (1.+kp2Lp3L/orbits->armlength)
    cmplx factorcexp31 = gcmplx::exp(I * prefactor * (1. + kp3Lp1L / orbits->armlength)); // prefactor * (1.+kp3Lp1L/orbits->armlength)

    cmplx factorsinc12 = d_sinc(prefactor * (1. - kn3));
    cmplx factorsinc21 = d_sinc(prefactor * (1. + kn3));
    cmplx factorsinc23 = d_sinc(prefactor * (1. - kn1));
    cmplx factorsinc32 = d_sinc(prefactor * (1. + kn1));
    cmplx factorsinc31 = d_sinc(prefactor * (1. - kn2));
    cmplx factorsinc13 = d_sinc(prefactor * (1. + kn2));

    // # Compute the Gslr - either scalars or vectors
    d_Gslr_holder Gslr_out;

    cmplx commonfac = I * prefactor * factorcexp0;
    Gslr_out.G12 = commonfac * n3Hn3 * factorsinc12 * factorcexp12;
    Gslr_out.G21 = commonfac * n3Hn3 * factorsinc21 * factorcexp12;
    Gslr_out.G23 = commonfac * n1Hn1 * factorsinc23 * factorcexp23;
    Gslr_out.G32 = commonfac * n1Hn1 * factorsinc32 * factorcexp23;
    Gslr_out.G31 = commonfac * n2Hn2 * factorsinc31 * factorcexp31;
    Gslr_out.G13 = commonfac * n2Hn2 * factorsinc13 * factorcexp31;

    return Gslr_out;
}

CUDA_CALLABLE_MEMBER
d_transferL_holder d_TDICombinationFD(d_Gslr_holder Gslr, double f, int TDItag, bool rescaled, bool tdi2, Orbits *orbits)
{
    // int TDItag == 1 is XYZ int TDItag == 2 is AET
    // int rescaled == 1 is True int rescaled == 0 is False
    d_transferL_holder transferL;
    cmplx factor, factorAE, factorT;
    cmplx tdi2_factor;
    cmplx I(0.0, 1.0);
    double x = PI * f * orbits->armlength / C_SI;
    cmplx z = gcmplx::exp(I * 2. * x);
    cmplx Xraw, Yraw, Zraw, Araw, Eraw, Traw;
    cmplx factor_convention, point5, c_one, c_two;
    if (TDItag == 1)
    {
        // # First-generation TDI XYZ
        // # With x=pifL, factor scaled out: 2I*sin2x*e2ix
        if (rescaled == 1)
            factor = 1.;
        else
            factor = 2. * I * sin(2. * x) * z;
        Xraw = Gslr.G21 + z * Gslr.G12 - Gslr.G31 - z * Gslr.G13;
        Yraw = Gslr.G32 + z * Gslr.G23 - Gslr.G12 - z * Gslr.G21;
        Zraw = Gslr.G13 + z * Gslr.G31 - Gslr.G23 - z * Gslr.G32;
        transferL.transferL1 = factor * Xraw;
        transferL.transferL2 = factor * Yraw;
        transferL.transferL3 = factor * Zraw;
        return transferL;
    }

    else
    {
        // # First-generation TDI AET from X,Y,Z
        // # With x=pifL, factors scaled out: A,E:I*SQRT2*sin2x*e2ix T:2*SQRT2*sin2x*sinx*e3ix
        // # Here we include a factor 2, because the code was first written using the definitions (2) of McWilliams&al_0911 where A,E,T are 1/2 of their LDC definitions
        factor_convention = cmplx(2., 0.0);
        if (rescaled == 1)
        {
            factorAE = cmplx(1., 0.0);
            factorT = cmplx(1., 0.0);
        }
        else
        {
            factorAE = I * SQRT2 * sin(2. * x) * z;
            factorT = 2. * SQRT2 * sin(2. * x) * sin(x) * gcmplx::exp(I * 3. * x);
        }

        Araw = 0.5 * ((1. + z) * (Gslr.G31 + Gslr.G13) - Gslr.G23 - z * Gslr.G32 - Gslr.G21 - z * Gslr.G12);
        Eraw = 0.5 * INVSQRT3 * ((1. - z) * (Gslr.G13 - Gslr.G31) + (2. + z) * (Gslr.G12 - Gslr.G32) + (1. + 2. * z) * (Gslr.G21 - Gslr.G23));
        Traw = INVSQRT6 * (Gslr.G21 - Gslr.G12 + Gslr.G32 - Gslr.G23 + Gslr.G13 - Gslr.G31);

        if (tdi2)
        {
            tdi2_factor = (-2. * I * sin(4. * x) * gcmplx::exp(I * 4. * x));
        }
        else
        {
            tdi2_factor = 1.0;
        }

        transferL.transferL1 = tdi2_factor * factor_convention * factorAE * Araw;
        transferL.transferL2 = tdi2_factor * factor_convention * factorAE * Eraw;
        transferL.transferL3 = tdi2_factor * factor_convention * factorT * Traw;
        return transferL;
    }
}

CUDA_CALLABLE_MEMBER
d_transferL_holder d_JustLISAFDresponseTDI(cmplx *H, double f, double t, double lam, double beta, int TDItag, bool rescaled, bool tdi2, int order_fresnel_stencil, Orbits *orbits)
{

    // funck
    CUDA_SHARED double kvec_all[3];
    double *kvec = &kvec_all[0];

#ifdef __HIPCC__
    CUDA_SHARED double p0_all[NUM_THREADS_RESPONSE * 3];
    double *p0 = &p0_all[threadIdx.x * 3];
#else
    double p0_all[3];
    double *p0 = &p0_all[0];
#endif
    kvec[0] = -cos(beta) * cos(lam);
    kvec[1] = -cos(beta) * sin(lam);
    kvec[2] = -sin(beta);

    // funcp0
    // TODO: fix
    double alpha = Omega0 * t;
    double c = cos(alpha);
    double s = sin(alpha);
    double a = AU_SI;

    p0[0] = a * c;
    p0[1] = a * s;
    p0[2] = 0. * t;

    // dot kvec with p0
    double kR = d_dot_product_1d(kvec, p0);

    double phaseRdelay = 2. * PI / C_SI * f * kR;

    // going to assume order_fresnel_stencil == 0 for now
    d_Gslr_holder Gslr = d_EvaluateGslr(t, f, H, kvec, 1, p0, orbits); // assumes full response
    d_Gslr_holder Tslr;                                                // use same struct because its the same setup
    cmplx m_I(0.0, -1.0);                                              // -1.0 -> mu_I

    // fill Tslr
    Tslr.G12 = Gslr.G12 * gcmplx::exp(m_I * phaseRdelay); // really -I*
    Tslr.G21 = Gslr.G21 * gcmplx::exp(m_I * phaseRdelay);
    Tslr.G23 = Gslr.G23 * gcmplx::exp(m_I * phaseRdelay);
    Tslr.G32 = Gslr.G32 * gcmplx::exp(m_I * phaseRdelay);
    Tslr.G31 = Gslr.G31 * gcmplx::exp(m_I * phaseRdelay);
    Tslr.G13 = Gslr.G13 * gcmplx::exp(m_I * phaseRdelay);

    d_transferL_holder transferL = d_TDICombinationFD(Tslr, f, TDItag, rescaled, tdi2, orbits);
    transferL.phaseRdelay = phaseRdelay;
    return transferL;
}

/**
 * Michael Katz added this function.
 * internal function that filles amplitude and phase for a specific frequency and mode.
 */
CUDA_CALLABLE_MEMBER
void response_modes(double *phases, double *response_out, int binNum, int mode_i, double *tf, double *freqs, double phi_ref, int ell, int mm, int length, int numBinAll, int numModes,
                    cmplx *H, double lam, double beta, int TDItag, bool rescaled, bool tdi2, int order_fresnel_stencil, Orbits *orbits)
{

    double eps = 1e-9;

    int start, increment;
#ifdef __HIPCC__
    start = threadIdx.x;
    increment = blockDim.x;
#else
    start = 0;
    increment = 1;
// #pragma omp parallel for
#endif
    for (int i = start; i < length; i += increment)
    {
        int mode_index = (binNum * numModes + mode_i) * length + i;
        int freq_index = binNum * length + i;

        double freq = freqs[mode_index];

        double t_wave_frame = tf[mode_index];

        d_transferL_holder transferL = d_JustLISAFDresponseTDI(H, freq, t_wave_frame, lam, beta, TDItag, rescaled, tdi2, order_fresnel_stencil, orbits);

        // transferL1_re
        int start_ind = 0 * numBinAll * numModes * length;
        int start_ind_old = start_ind;
        response_out[start_ind + mode_index] = gcmplx::real(transferL.transferL1);

        // transferL1_im
        start_ind = 1 * numBinAll * numModes * length;
        response_out[start_ind + mode_index] = gcmplx::imag(transferL.transferL1);

        // transferL1_re
        start_ind = 2 * numBinAll * numModes * length;
        response_out[start_ind + mode_index] = gcmplx::real(transferL.transferL2);

        // transferL1_re
        start_ind = 3 * numBinAll * numModes * length;
        response_out[start_ind + mode_index] = gcmplx::imag(transferL.transferL2);

        // transferL1_re
        start_ind = 4 * numBinAll * numModes * length;
        response_out[start_ind + mode_index] = gcmplx::real(transferL.transferL3);

        // transferL1_re
        start_ind = 5 * numBinAll * numModes * length;
        response_out[start_ind + mode_index] = gcmplx::imag(transferL.transferL3);

        double phase_change = transferL.phaseRdelay;

        // adjust phase
        phases[mode_index] += phase_change;
    }
}

/*
Calculate spin weighted spherical harmonics
*/
CUDA_CALLABLE_MEMBER
cmplx SpinWeightedSphericalHarmonic(int s, int l, int m, double theta, double phi)
{
    // l=2
    double fac;
    if ((l == 2) && (m == -2))
        fac = sqrt(5.0 / (64.0 * PI)) * (1.0 - cos(theta)) * (1.0 - cos(theta));
    else if ((l == 2) && (m == -1))
        fac = sqrt(5.0 / (16.0 * PI)) * sin(theta) * (1.0 - cos(theta));
    else if ((l == 2) && (m == 1))
        fac = sqrt(5.0 / (16.0 * PI)) * sin(theta) * (1.0 + cos(theta));
    else if ((l == 2) && (m == 2))
        fac = sqrt(5.0 / (64.0 * PI)) * (1.0 + cos(theta)) * (1.0 + cos(theta));
    // l=3
    else if ((l == 3) && (m == -3))
        fac = sqrt(21.0 / (2.0 * PI)) * cos(theta / 2.0) * pow(sin(theta / 2.0), 5.0);
    else if ((l == 3) && (m == -2))
        fac = sqrt(7.0 / (4.0 * PI)) * (2.0 + 3.0 * cos(theta)) * pow(sin(theta / 2.0), 4.0);
    else if ((l == 3) && (m == 2))
        fac = sqrt(7.0 / PI) * pow(cos(theta / 2.0), 4.0) * (-2.0 + 3.0 * cos(theta)) / 2.0;
    else if ((l == 3) && (m == 3))
        fac = -sqrt(21.0 / (2.0 * PI)) * pow(cos(theta / 2.0), 5.0) * sin(theta / 2.0);
    // l=4
    else if ((l == 4) && (m == -4))
        fac = 3.0 * sqrt(7.0 / PI) * pow(cos(theta / 2.0), 2.0) * pow(sin(theta / 2.0), 6.0);
    else if ((l == 4) && (m == -3))
        fac = 3.0 * sqrt(7.0 / (2.0 * PI)) * cos(theta / 2.0) * (1.0 + 2.0 * cos(theta)) * pow(sin(theta / 2.0), 5.0);

    else if ((l == 4) && (m == 3))
        fac = -3.0 * sqrt(7.0 / (2.0 * PI)) * pow(cos(theta / 2.0), 5.0) * (-1.0 + 2.0 * cos(theta)) * sin(theta / 2.0);
    else if ((l == 4) && (m == 4))
        fac = 3.0 * sqrt(7.0 / PI) * pow(cos(theta / 2.0), 6.0) * pow(sin(theta / 2.0), 2.0);

    // Result
    cmplx I(0.0, 1.0);
    if (m == 0)
        return cmplx(fac, 0.0);
    else
    {
        cmplx phaseTerm(m * phi, 0.0);
        return fac * exp(I * phaseTerm);
    }
}

/*
custom dot product in 2d
*/
CUDA_CALLABLE_MEMBER
void dot_product_2d(double *out, double *arr1, int m1, int n1, double *arr2, int m2, int n2)
{

    // dev and stride are on output
    for (int i = 0; i < m1; i++)
    {
        for (int j = 0; j < n2; j++)
        {
            out[(i * 3 + j)] = 0.0;
            for (int k = 0; k < n1; k++)
            {
                out[(i * 3 + j)] += arr1[i * 3 + k] * arr2[k * 3 + j];
            }
        }
    }
}

/*
Custom dot product in 1d
*/
CUDA_CALLABLE_MEMBER
double dot_product_1d(double arr1[3], double arr2[3])
{
    double out = 0.0;
    for (int i = 0; i < 3; i++)
    {
        out += arr1[i] * arr2[i];
    }
    return out;
}

/**
 * Michael Katz added this function.
 * Main function for calculating PhenomHM in the form used by Michael Katz
 * This is setup to allow for pre-allocation of arrays. Therefore, all arrays
 * should be setup outside of this function.
 */
CUDA_CALLABLE_MEMBER
void responseCore(
    double *phases,
    double *response_out,
    int *ells,
    int *mms,
    double *tf,
    double *freqs,        /**< GW frequecny list [Hz] */
    const double phi_ref, /**< orbital phase at f_ref */
    double inc,
    double lam,
    double beta,
    double psi,
    int length, /**< reference GW frequency */
    int numModes,
    int binNum,
    int numBinAll,
    int TDItag, bool rescaled, bool tdi2, int order_fresnel_stencil, Orbits *orbits)
{

    int ell, mm;

    //// setup response
    CUDA_SHARED double HSplus[9];
    CUDA_SHARED double HScross[9];

    CUDA_SHARED cmplx H_mat[3 * 3];
    CUDA_SHARED double Hplus[3 * 3];
    CUDA_SHARED double Hcross[3 * 3];
    CUDA_SHARED double kvec[3];
    CUDA_SHARED double O1[3 * 3];
    CUDA_SHARED double invO1[3 * 3];
    CUDA_SHARED double out1[3 * 3];

    if THREAD_ZERO
    {
        HSplus[0] = 1.;
        HSplus[1] = 0.;
        HSplus[2] = 0.;
        HSplus[3] = 0.;
        HSplus[4] = -1.;
        HSplus[5] = 0.;
        HSplus[6] = 0.;
        HSplus[7] = 0.;
        HSplus[8] = 0.;

        HScross[0] = 0.;
        HScross[1] = 1.;
        HScross[2] = 0.;
        HScross[3] = 1.;
        HScross[4] = 0.;
        HScross[5] = 0.;
        HScross[6] = 0.;
        HScross[7] = 0.;
        HScross[8] = 0.;

        // ##### Based on the f-n by Sylvain   #####

        // Wave unit vector

        // double* kvec = &kvec_all[threadIdx.x * 3];
        kvec[0] = -cos(beta) * cos(lam);
        kvec[1] = -cos(beta) * sin(lam);
        kvec[2] = -sin(beta);

        // Compute constant matrices Hplus and Hcross in the SSB frame
        double clambd = cos(lam);
        double slambd = sin(lam);
        double cbeta = cos(beta);
        double sbeta = sin(beta);
        double cpsi = cos(psi);
        double spsi = sin(psi);

        O1[0] = cpsi * slambd - clambd * sbeta * spsi;
        O1[1] = -clambd * cpsi * sbeta - slambd * spsi;
        O1[2] = -cbeta * clambd;
        O1[3] = -clambd * cpsi - sbeta * slambd * spsi;
        O1[4] = -cpsi * sbeta * slambd + clambd * spsi;
        O1[5] = -cbeta * slambd;
        O1[6] = cbeta * spsi;
        O1[7] = cbeta * cpsi;
        O1[8] = -sbeta;

        invO1[0] = cpsi * slambd - clambd * sbeta * spsi;
        invO1[1] = -clambd * cpsi - sbeta * slambd * spsi;
        invO1[2] = cbeta * spsi;
        invO1[3] = -clambd * cpsi * sbeta - slambd * spsi;
        invO1[4] = -cpsi * sbeta * slambd + clambd * spsi;
        invO1[5] = cbeta * cpsi;
        invO1[6] = -cbeta * clambd;
        invO1[7] = -cbeta * slambd;
        invO1[8] = -sbeta;

        // get Hplus
        dot_product_2d(out1, HSplus, 3, 3, invO1, 3, 3);

        dot_product_2d(Hplus, O1, 3, 3, out1, 3, 3);

        // get Hcross
        dot_product_2d(out1, HScross, 3, 3, invO1, 3, 3);
        dot_product_2d(Hcross, O1, 3, 3, out1, 3, 3);
    }
    CUDA_SYNC_THREADS;
    cmplx I = cmplx(0.0, 1.0);
    cmplx Ylm, Yl_m, Yfactorplus, Yfactorcross;

    double trans1, trans2;
    for (int mode_i = 0; mode_i < numModes; mode_i++)
    {

        ell = ells[mode_i];
        mm = mms[mode_i];

        if THREAD_ZERO
        {
            Ylm = SpinWeightedSphericalHarmonic(-2, ell, mm, inc, phi_ref);
            Yl_m = pow(-1.0, ell) * gcmplx::conj(SpinWeightedSphericalHarmonic(-2, ell, -1 * mm, inc, phi_ref));
            Yfactorplus = 1. / 2 * (Ylm + Yl_m);
            // # Yfactorcross = 1j/2 * (Y22 - Y2m2)  ### SB, should be for correct phase conventions
            Yfactorcross = 1. / 2. * I * (Ylm - Yl_m); //  ### SB, minus because the phase convention is opposite, we'll tace c.c. at the end
            // # Yfactorcross = -1j/2 * (Y22 - Y2m2)  ### SB, minus because the phase convention is opposite, we'll tace c.c. at the end
            // # Yfactorcross = 1j/2 * (Y22 - Y2m2)  ### SB, minus because the phase convention is opposite, we'll tace c.c. at the end
            // # The matrix H_mat is now complex

            // # H_mat = np.conjugate((Yfactorplus*Hplus + Yfactorcross*Hcross))  ### SB: H_ij = H_mat A_22 exp(i\Psi(f))
            for (int i = 0; i < 3; i++)
            {
                for (int j = 0; j < 3; j++)
                {
                    trans1 = Hplus[(i * 3 + j)];
                    trans2 = Hcross[(i * 3 + j)];
                    H_mat[(i * 3 + j)] = (Yfactorplus * trans1 + Yfactorcross * trans2);
                }
            }
        }
        CUDA_SYNC_THREADS;

        // if (threadIdx.x == 0) printf("CHECK: %.18e %.18e %.18e\n", inc, phi_ref, psi);
        response_modes(phases, response_out, binNum, mode_i, tf, freqs, phi_ref, ell, mm, length, numBinAll, numModes,
                       H_mat, lam, beta, TDItag, rescaled, tdi2, order_fresnel_stencil, orbits);
    }
}

////////////
// response
////////////

#define MAX_MODES 6

CUDA_KERNEL
void response(
    double *phases,
    double *response_out,
    double *tf,
    int *ells_in,
    int *mms_in,
    double *freqs,   /**< Frequency points at which to evaluate the waveform (Hz) */
    double *phi_ref, /**< reference orbital phase (rad) */
    double *inc,
    double *lam,
    double *beta,
    double *psi,
    int TDItag, bool rescaled, bool tdi2, int order_fresnel_stencil,
    int numModes,
    int length,
    int numBinAll,
    Orbits *orbits)
{

    CUDA_SHARED int ells[MAX_MODES];
    CUDA_SHARED int mms[MAX_MODES];

    int start, increment;
#ifdef __HIPCC__
    start = threadIdx.x;
    increment = blockDim.x;
#else
    start = 0;
    increment = 1;
// #pragma omp parallel for
#endif
    for (int i = start; i < numModes; i += increment)
    {
        ells[i] = ells_in[i];
        mms[i] = mms_in[i];
    }

    CUDA_SYNC_THREADS;

#ifdef __HIPCC__
    start = blockIdx.x;
    increment = gridDim.x;
#else
    start = 0;
    increment = 1;
// #pragma omp parallel for
#endif
    for (int binNum = start; binNum < numBinAll; binNum += increment)
    {
        responseCore(phases, response_out, ells, mms, tf, freqs, phi_ref[binNum], inc[binNum], lam[binNum], beta[binNum], psi[binNum], length, numModes, binNum, numBinAll,
                     TDItag, rescaled, tdi2, order_fresnel_stencil, orbits);
    }
}

void LISA_response(
    double *response_out,
    int *ells_in,
    int *mms_in,
    double *freqs,   /**< Frequency points at which to evaluate the waveform (Hz) */
    double *phi_ref, /**< reference orbital phase (rad) */
    double *inc,
    double *lam,
    double *beta,
    double *psi,
    int TDItag, bool rescaled, bool tdi2, int order_fresnel_stencil,
    int numModes,
    int length,
    int numBinAll,
    int includesAmps, Orbits *orbits)
{

    int start_param = includesAmps; // if it has amps, start_param is 1, else 0

    // get arrays out of main holder
    double *phases = &response_out[start_param * numBinAll * numModes * length];
    double *tf = &response_out[(start_param + 1) * numBinAll * numModes * length];
    double *response_vals = &response_out[(start_param + 2) * numBinAll * numModes * length];

#ifdef __HIPCC__

    int nblocks2 = numBinAll;

    // copy self to GPU
    Orbits *orbits_gpu;
    gpuErrchk(hipMalloc(&orbits_gpu, sizeof(Orbits)));
    gpuErrchk(hipMemcpy(orbits_gpu, orbits, sizeof(Orbits), hipMemcpyHostToDevice));

    // put each binary on its own block

    response<<<nblocks2, NUM_THREADS_RESPONSE>>>(
        phases,
        response_vals,
        tf,
        ells_in,
        mms_in,
        freqs,   /**< Frequency points at which to evaluate the waveform (Hz) */
        phi_ref, /**< reference orbital phase (rad) */
        inc,
        lam,
        beta,
        psi,
        TDItag, rescaled, tdi2, order_fresnel_stencil,
        numModes,
        length,
        numBinAll, orbits_gpu);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipFree(orbits_gpu));

#else
    response(
        phases,
        response_vals,
        tf,
        ells_in,
        mms_in,
        freqs,   /**< Frequency points at which to evaluate the waveform (Hz) */
        phi_ref, /**< reference orbital phase (rad) */
        inc,
        lam,
        beta,
        psi,
        TDItag, rescaled, tdi2, order_fresnel_stencil,
        numModes,
        length,
        numBinAll,
        orbits);
#endif
}
