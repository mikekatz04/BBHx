#include "hip/hip_runtime.h"
#include "constants.h"
#include "global.h"
#include "WaveformBuild.hh"

#define NUM_THREADS_BUILD 256

#ifdef __HIPCC__
__device__ double atomicAddDouble(double *address, double val)
{
    unsigned long long *address_as_ull =
        (unsigned long long *)address;
    unsigned long long old = *address_as_ull, assumed;

    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ void atomicAddComplex(cmplx *a, cmplx b)
{
    // transform the addresses of real and imag. parts to double pointers
    double *x = (double *)a;
    double *y = x + 1;
    // use atomicAdd for double variables
    atomicAddDouble(x, b.real());
    atomicAddDouble(y, b.imag());
}
#endif

#define DATA_BLOCK 128
#define NUM_INTERPS 9

CUDA_CALLABLE_MEMBER
cmplx get_ampphasefactor(double amp, double phase, double phaseShift)
{
    return amp * gcmplx::exp(cmplx(0.0, phase + phaseShift));
}

CUDA_CALLABLE_MEMBER
void combine_information(cmplx *channel1, cmplx *channel2, cmplx *channel3, double amp, double phase, double tf, cmplx transferL1, cmplx transferL2, cmplx transferL3, double t_start, double t_end)
{
    if (((tf >= t_start)) && ((tf <= t_end) || (t_end <= 0.0)))
    {
        // this is the final waveform combination
        // only happens if it is in the time bounds
        cmplx amp_phase_term = amp * gcmplx::exp(cmplx(0.0, phase));

        *channel1 = gcmplx::conj(transferL1 * amp_phase_term);
        *channel2 = gcmplx::conj(transferL2 * amp_phase_term);
        *channel3 = gcmplx::conj(transferL3 * amp_phase_term);
    }
}

#define NUM_TERMS 4

#define MAX_NUM_COEFF_TERMS 1200

// interpolate to TDI channels
CUDA_KERNEL
void TDI(cmplx *templateChannels, double *dataFreqsIn, double *freqsOld, double *propArrays, double *c1In, double *c2In, double *c3In, int old_length, int data_length, int numBinAll, int numModes, double t_obs_start, double t_obs_end, int *inds, int ind_start, int ind_length, int bin_i)
{

    int start, increment;
#ifdef __HIPCC__
    start = blockIdx.x * blockDim.x + threadIdx.x;
    increment = blockDim.x * gridDim.x;
#else
    start = 0;
    increment = 1;
// #pragma omp parallel for
#endif
    for (int i = start; i < ind_length; i += increment)
    {
        // get x information for this spline evaluation
        double f = dataFreqsIn[i + ind_start];

        cmplx trans_complex1 = 0.0;
        cmplx trans_complex2 = 0.0;
        cmplx trans_complex3 = 0.0;

        for (int mode_i = 0; mode_i < numModes; mode_i += 1)
        {
            int ind_here = inds[mode_i * ind_length + i];
            if ((ind_here <= -1) || (ind_here >= old_length - 1))
                continue;
            // evaluate all spline quantities
            double f_old = freqsOld[(bin_i * numModes + mode_i) * old_length + ind_here];
            double f_old1 = freqsOld[(bin_i * numModes + mode_i) * old_length + ind_here + 1];

            double x = f - f_old;
            double x2 = x * x;
            double x3 = x * x2;

            int int_shared = ((0 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
            double amp = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

            int_shared = ((1 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
            double phase = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

            int_shared = ((2 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
            double tf = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

            int_shared = ((3 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
            double transferL1_re = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

            int_shared = ((4 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
            double transferL1_im = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

            int_shared = ((5 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
            double transferL2_re = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

            int_shared = ((6 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
            double transferL2_im = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

            int_shared = ((7 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
            double transferL3_re = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

            int_shared = ((8 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
            double transferL3_im = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

            cmplx channel1(0.0, 0.0);
            cmplx channel2(0.0, 0.0);
            cmplx channel3(0.0, 0.0);

            combine_information(&channel1, &channel2, &channel3, amp, phase, tf, cmplx(transferL1_re, transferL1_im), cmplx(transferL2_re, transferL2_im), cmplx(transferL3_re, transferL3_im), t_obs_start, t_obs_end);

            // add all modes together directly
            trans_complex1 += channel1;
            trans_complex2 += channel2;
            trans_complex3 += channel3;
        }

        templateChannels[0 * ind_length + i] = trans_complex1;
        templateChannels[1 * ind_length + i] = trans_complex2;
        templateChannels[2 * ind_length + i] = trans_complex3;
    }
}

void InterpTDI(long *templateChannels_ptrs, double *dataFreqs, double *freqs, double *propArrays, double *c1, double *c2, double *c3, double *t_start_in, double *t_end_in, int length, int data_length, int numBinAll, int numModes, long *inds_ptrs, int *inds_start, int *ind_lengths)
{
#ifdef __HIPCC__
    hipStream_t streams[numBinAll];
#endif

    // interpolation is done in streams on GPU
    // #pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        // get all information ready included casting pointers properly
        int length_bin_i = ind_lengths[bin_i];
        int ind_start = inds_start[bin_i];
        int *inds = (int *)inds_ptrs[bin_i];

        double t_start = t_start_in[bin_i];
        double t_end = t_end_in[bin_i];

        cmplx *templateChannels = (cmplx *)templateChannels_ptrs[bin_i];

        int nblocks3 = std::ceil((length_bin_i + NUM_THREADS_BUILD - 1) / NUM_THREADS_BUILD);

#ifdef __HIPCC__
        dim3 gridDim(nblocks3, 1);
        hipStreamCreate(&streams[bin_i]);
        TDI<<<gridDim, NUM_THREADS_BUILD, 0, streams[bin_i]>>>(templateChannels, dataFreqs, freqs, propArrays, c1, c2, c3, length, data_length, numBinAll, numModes, t_start, t_end, inds, ind_start, length_bin_i, bin_i);
#else
        TDI(templateChannels, dataFreqs, freqs, propArrays, c1, c2, c3, length, data_length, numBinAll, numModes, t_start, t_end, inds, ind_start, length_bin_i, bin_i);
#endif
    }

#ifdef __HIPCC__
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    // #pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        // destroy the streams
        hipStreamDestroy(streams[bin_i]);
    }
#endif
}

// directly fill waveform with no interpolation
// parallel method here is one block per binary
CUDA_KERNEL
void fill_waveform(cmplx *templateChannels,
                   double *bbh_buffer,
                   int numBinAll, int data_length, int nChannels, int numModes, double *t_start, double *t_end)
{

    cmplx I(0.0, 1.0);

    cmplx temp_channel1 = 0.0, temp_channel2 = 0.0, temp_channel3 = 0.0;
    int start, increment;
#ifdef __HIPCC__
    start = blockIdx.x;
    increment = gridDim.x;
#else
    start = 0;
    increment = 1;
// #pragma omp parallel for
#endif
    for (int bin_i = start; bin_i < numBinAll; bin_i += increment)
    {

        double t_start_bin = t_start[bin_i];
        double t_end_bin = t_end[bin_i];

        int start2, increment2;
#ifdef __HIPCC__
        start2 = threadIdx.x;
        increment2 = blockDim.x;
#else
        start2 = 0;
        increment2 = 1;
// #pragma omp parallel for
#endif
        for (int i = start2; i < data_length; i += increment2)
        {
            cmplx temp_channel1 = 0.0;
            cmplx temp_channel2 = 0.0;
            cmplx temp_channel3 = 0.0;
            for (int mode_i = 0; mode_i < numModes; mode_i += 1)
            {

                // get each value directly out of the holder arrays

                int ind = ((0 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double amp = bbh_buffer[ind];

                ind = ((1 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double phase = bbh_buffer[ind];

                ind = ((2 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double tf = bbh_buffer[ind];

                ind = ((3 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL1_re = bbh_buffer[ind];

                ind = ((4 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL1_im = bbh_buffer[ind];

                ind = ((5 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL2_re = bbh_buffer[ind];

                ind = ((6 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL2_im = bbh_buffer[ind];

                ind = ((7 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL3_re = bbh_buffer[ind];

                ind = ((8 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL3_im = bbh_buffer[ind];

                cmplx channel1 = 0.0 + 0.0 * I;
                cmplx channel2 = 0.0 + 0.0 * I;
                cmplx channel3 = 0.0 + 0.0 * I;

                combine_information(&channel1, &channel2, &channel3, amp, phase, tf, cmplx(transferL1_re, transferL1_im), cmplx(transferL2_re, transferL2_im), cmplx(transferL3_re, transferL3_im), t_start_bin, t_end_bin);

                temp_channel1 += channel1;
                temp_channel2 += channel2;
                temp_channel3 += channel3;
            }

            templateChannels[(bin_i * 3 + 0) * data_length + i] = temp_channel1;
            templateChannels[(bin_i * 3 + 1) * data_length + i] = temp_channel2;
            templateChannels[(bin_i * 3 + 2) * data_length + i] = temp_channel3;
        }
    }
}

void direct_sum(cmplx *templateChannels,
                double *bbh_buffer,
                int numBinAll, int data_length, int nChannels, int numModes, double *t_start, double *t_end)
{

    // block per binary
    int nblocks5 = numBinAll;

#ifdef __HIPCC__
    fill_waveform<<<nblocks5, NUM_THREADS_BUILD>>>(templateChannels, bbh_buffer, numBinAll, data_length, nChannels, numModes, t_start, t_end);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
#else
    fill_waveform(templateChannels, bbh_buffer, numBinAll, data_length, nChannels, numModes, t_start, t_end);
#endif
}
